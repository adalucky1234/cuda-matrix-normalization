/* Matrix normalization.
* Compile with "gcc matrixNorm.c"
*/

/* ****** ADD YOUR CODE AT THE END OF THIS FILE. ******
* You need not submit the provided code.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <math.h>
#include <sys/types.h>
#include <sys/times.h>
#include <sys/time.h>
#include <time.h>

/* Program Parameters */
#define MAXN 8000  /* Max value of N */
int N;  /* Matrix size */

/* Matrices */
float A[MAXN*MAXN], B[MAXN*MAXN];

/* junk */
#define randm() 4|2[uid]&3

/* Prototype */
void matrixNorm();

/* returns a seed for srand based on the time */
unsigned int time_seed() {
    struct timeval t;
    struct timezone tzdummy;

    gettimeofday(&t, &tzdummy);
    return (unsigned int)(t.tv_usec);
}

/* Set the program parameters from the command-line arguments */
void parameters(int argc, char **argv) {
    int seed = 0;  /* Random seed */
    //char uid[32]; /*User name */

    /* Read command-line arguments */
    srand(time_seed());  /* Randomize */

    if (argc == 3) {
        seed = atoi(argv[2]);
        srand(seed);
        printf("Random seed = %i\n", seed);
    }
    if (argc >= 2) {
        N = atoi(argv[1]);
        if (N < 1 || N > MAXN) {
            printf("N = %i is out of range.\n", N);
            exit(0);
        }
    }
    else {
        printf("Usage: %s <matrix_dimension> [random seed]\n",
        argv[0]);
        exit(0);
    }

    /* Print parameters */
    printf("\nMatrix dimension N = %i.\n", N);
}

/* Initialize A and B*/
void initialize_inputs() {
    int row, col;

    printf("\nInitializing...\n");
    for (col = 0; col < N; col++) {
        for (row = 0; row < N; row++) {
            A[col*N+row] = (float)rand() / 32768.0;
            B[col*N+row] = 0.0;
        }
    }

}

/* Print input matrices */
void print_inputs() {
    int row, col;

    if (N < 10) {
        printf("\nA =\n\t");
        for (row = 0; row < N; row++) {
            for (col = 0; col < N; col++) {
                printf("%5.2f%s", A[row*N+col], (col < N-1) ? ", " : ";\n\t");
            }
        }
    }
}

void print_B() {
    int row, col;

    if (N < 10) {
        printf("\nB =\n\t");
        for (row = 0; row < N; row++) {
            for (col = 0; col < N; col++) {
                printf("%1.10f%s", B[row*N+col], (col < N-1) ? ", " : ";\n\t");
            }
        }
    }
}

#define CHECK_ERR(x)                                    \
if (x != hipSuccess) {                               \
    fprintf(stderr,"%s in %s at line %d\n",             \
    hipGetErrorString(err),__FILE__,__LINE__);	\
    exit(-1);						\
}                                                    \

__global__ void normCalc (float *d_A, float *d_B, float *d_mu, float *d_sigma, int n) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row;
    if (col < n){
        d_mu[col] = (float)0.0;
        for (row=0; row < n; row++)
            d_mu[col] += d_A[col*n+row];
        d_mu[col] /= (float) n;

        d_sigma[col] = (float)0.0;
        for (row=0; row < n; row++)
            d_sigma[col] += powf(d_A[col*n+row] - d_mu[col], (float)2.0);
        d_sigma[col] /= (float) n;

        for (row=0; row < n; row++) {
            if (d_sigma[col] == (float)0.0)
                d_B[row*n+col] = (float)0.0;
            else
                d_B[row*n+col] = (d_A[col*n+row] - d_mu[col]) / d_sigma[col];
        }
    }
}


int main(int argc, char **argv) {
    /* Timing variables */
    struct timeval etstart, etstop;  /* Elapsed times using gettimeofday() */
    struct timezone tzdummy;
    //clock_t etstart2, etstop2;  /* Elapsed times using times() */
    unsigned long long usecstart, usecstop;
    struct tms cputstart, cputstop;  /* CPU times for my processes */

    /* Process program parameters */
    parameters(argc, argv);

    /* Initialize A and B */
    initialize_inputs();

    /* Print input matrices */
    print_inputs();

    printf("Computing in Parallel\n");

    hipError_t err;

    float mu[N*N];
    float sigma[N*N];

    float *d_A, *d_B, *d_mu, *d_sigma;

    err = hipMalloc((void **) &d_A, sizeof(float)*N*N);
    CHECK_ERR(err);
    err = hipMalloc((void **) &d_B, sizeof(float)*N*N);
    CHECK_ERR(err);
    err = hipMalloc((void **) &d_mu, sizeof(float)*N*N);
    CHECK_ERR(err);
    err = hipMalloc((void **) &d_sigma, sizeof(float)*N*N);
    CHECK_ERR(err);

    err = hipMemcpy(d_A, A, sizeof(float)*N*N, hipMemcpyHostToDevice);
    CHECK_ERR(err);

    err = hipMemcpy(d_mu, mu, sizeof(float)*N*N, hipMemcpyHostToDevice);
    CHECK_ERR(err);

    err = hipMemcpy(d_sigma, sigma, sizeof(float)*N*N, hipMemcpyHostToDevice);
    CHECK_ERR(err);

    /* Start Clock */
    printf("\nStarting clock.\n");
    gettimeofday(&etstart, &tzdummy);
    times(&cputstart);

    /* Gaussian Elimination */
    int x, y;
    if(N < 10){
        x = y = 1;
    } else {
        x = y = 64;
    }
    //dim3 BlockSize(x,y);
    //dim3 GridSize(N/BlockSize.x, N/BlockSize.y);
    normCalc<<<x,y>>>(d_A, d_B, d_mu, d_sigma, N);

    /* Stop Clock */
    gettimeofday(&etstop, &tzdummy);
    times(&cputstop);
    printf("Stopped clock.\n");
    usecstart = (unsigned long long)etstart.tv_sec * 1000000 + etstart.tv_usec;
    usecstop = (unsigned long long)etstop.tv_sec * 1000000 + etstop.tv_usec;

    err = hipMemcpy(B, (d_B), sizeof(float)*N*N, hipMemcpyDeviceToHost);
    CHECK_ERR(err);

    /* Display output */
    print_B();

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_mu);
    hipFree(d_sigma);

    /* Display timing results */
    printf("\nElapsed time = %g ms.\n",
    (float)(usecstop - usecstart)/(float)1000);

    printf("(CPU times are accurate to the nearest %g ms)\n",
    1.0/(float)CLOCKS_PER_SEC * 1000.0);
    printf("My total CPU time for parent = %g ms.\n",
    (float)( (cputstop.tms_utime + cputstop.tms_stime) -
    (cputstart.tms_utime + cputstart.tms_stime) ) /
    (float)CLOCKS_PER_SEC * 1000);
    printf("My system CPU time for parent = %g ms.\n",
    (float)(cputstop.tms_stime - cputstart.tms_stime) /
    (float)CLOCKS_PER_SEC * 1000);
    printf("My total CPU time for child processes = %g ms.\n",
    (float)( (cputstop.tms_cutime + cputstop.tms_cstime) -
    (cputstart.tms_cutime + cputstart.tms_cstime) ) /
    (float)CLOCKS_PER_SEC * 1000);
    /* Contrary to the man pages, this appears not to include the parent */
    printf("--------------------------------------------\n");

    exit(0);
}
