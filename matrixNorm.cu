/* Matrix normalization.
 * Compile with "gcc matrixNorm.c"
 */

/* ****** ADD YOUR CODE AT THE END OF THIS FILE. ******
 * You need not submit the provided code.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <math.h>
#include <sys/types.h>
#include <sys/times.h>
#include <sys/time.h>
#include <time.h>

/* Program Parameters */
#define MAXN 8000  /* Max value of N */
int N;  /* Matrix size */

/* Matrices */
float A[MAXN][MAXN], B[MAXN][MAXN];

/* junk */
#define randm() 4|2[uid]&3

/* Prototype */
void matrixNorm();

/* returns a seed for srand based on the time */
unsigned int time_seed() {
  struct timeval t;
  struct timezone tzdummy;

  gettimeofday(&t, &tzdummy);
  return (unsigned int)(t.tv_usec);
}

/* Set the program parameters from the command-line arguments */
void parameters(int argc, char **argv) {
  int seed = 0;  /* Random seed */
  char uid[32]; /*User name */

  /* Read command-line arguments */
  srand(time_seed());  /* Randomize */

  if (argc == 3) {
    seed = atoi(argv[2]);
    srand(seed);
    printf("Random seed = %i\n", seed);
  }
  if (argc >= 2) {
    N = atoi(argv[1]);
    if (N < 1 || N > MAXN) {
      printf("N = %i is out of range.\n", N);
      exit(0);
    }
  }
  else {
    printf("Usage: %s <matrix_dimension> [random seed]\n",
           argv[0]);
    exit(0);
  }

  /* Print parameters */
  printf("\nMatrix dimension N = %i.\n", N);
}

/* Initialize A and B*/
void initialize_inputs() {
  int row, col;

  printf("\nInitializing...\n");
  for (col = 0; col < N; col++) {
    for (row = 0; row < N; row++) {
      A[row][col] = (float)rand() / 32768.0;
      B[row][col] = 0.0;
    }
  }

}

/* Print input matrices */
void print_inputs() {
  int row, col;

  if (N < 50) {
    printf("\nA =\n\t");
    for (row = 0; row < N; row++) {
      for (col = 0; col < N; col++) {
	    printf("%5.2f%s", A[row][col], (col < N-1) ? ", " : ";\n\t");
      }
    }
  }
}

void print_B() {
    int row, col;

    if (N < 50) {
        printf("\nB =\n\t");
        for (row = 0; row < N; row++) {
            for (col = 0; col < N; col++) {
                printf("%1.10f%s", B[row][col], (col < N-1) ? ", " : ";\n\t");
            }
        }
    }
}

int main(int argc, char **argv) {
  /* Timing variables */
  struct timeval etstart, etstop;  /* Elapsed times using gettimeofday() */
  struct timezone tzdummy;
  clock_t etstart2, etstop2;  /* Elapsed times using times() */
  unsigned long long usecstart, usecstop;
  struct tms cputstart, cputstop;  /* CPU times for my processes */

  /* Process program parameters */
  parameters(argc, argv);

  /* Initialize A and B */
  initialize_inputs();

  /* Print input matrices */
  print_inputs();

  /* Start Clock */
  printf("\nStarting clock.\n");
  gettimeofday(&etstart, &tzdummy);
  etstart2 = times(&cputstart);

  /* Gaussian Elimination */
  matrixNorm();

  /* Stop Clock */
  gettimeofday(&etstop, &tzdummy);
  etstop2 = times(&cputstop);
  printf("Stopped clock.\n");
  usecstart = (unsigned long long)etstart.tv_sec * 1000000 + etstart.tv_usec;
  usecstop = (unsigned long long)etstop.tv_sec * 1000000 + etstop.tv_usec;

  /* Display output */
  print_B();

  /* Display timing results */
  printf("\nElapsed time = %g ms.\n",
	 (float)(usecstop - usecstart)/(float)1000);

  printf("(CPU times are accurate to the nearest %g ms)\n",
	 1.0/(float)CLOCKS_PER_SEC * 1000.0);
  printf("My total CPU time for parent = %g ms.\n",
	 (float)( (cputstop.tms_utime + cputstop.tms_stime) -
		  (cputstart.tms_utime + cputstart.tms_stime) ) /
	 (float)CLOCKS_PER_SEC * 1000);
  printf("My system CPU time for parent = %g ms.\n",
	 (float)(cputstop.tms_stime - cputstart.tms_stime) /
	 (float)CLOCKS_PER_SEC * 1000);
  printf("My total CPU time for child processes = %g ms.\n",
	 (float)( (cputstop.tms_cutime + cputstop.tms_cstime) -
		  (cputstart.tms_cutime + cputstart.tms_cstime) ) /
	 (float)CLOCKS_PER_SEC * 1000);
      /* Contrary to the man pages, this appears not to include the parent */
  printf("--------------------------------------------\n");

  exit(0);
}

/* ------------------ Above Was Provided --------------------- */

/****** You will replace this routine with your own parallel version *******/
/* Provided global variables are MAXN, N, A[][] and B[][],
 * defined in the beginning of this code.  B[][] is initialized to zeros.
 */

 #define CHECK_ERR(x)                                    \
   if (x != hipSuccess) {                               \
     fprintf(stderr,"%s in %s at line %d\n",             \
 	    hipGetErrorString(err),__FILE__,__LINE__);	\
     exit(-1);						\
  }                                                    \

  __global__ void normCalc (float* d_A, float* d_B, float* d_mu, float* d_sigma, int n) {
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    if (col < n && row < n) {
        //for (row=0; row < n; row++)
            d_B[row*n+col] = d_A[row*n+col];
    }
}


void matrixNorm() {

    printf("Computing in Parallel\n");

    hipError_t err;

    float mu[N];
    float sigma[N];
    memset(mu, 0.0, sizeof(mu));
    memset(sigma, 0.0, sizeof(sigma));

    float *d_A, *d_B, *d_mu, *d_sigma;

    err = hipMalloc((void **) &d_A, sizeof(float)*N*N);
    CHECK_ERR(err);
    err = hipMalloc((void **) &d_B, sizeof(float)*N*N);
    CHECK_ERR(err);
    err = hipMalloc((void **) &d_mu, sizeof(float)*N*N);
    CHECK_ERR(err);
    err = hipMalloc((void **) &d_sigma, sizeof(float)*N*N);
    CHECK_ERR(err);


    err = hipMemcpy(d_A, A, sizeof(float)*N*N, hipMemcpyHostToDevice);
    CHECK_ERR(err);

    err = hipMemcpy(d_mu, mu, sizeof(float)*N*N, hipMemcpyHostToDevice);
    CHECK_ERR(err);

    err = hipMemcpy(d_sigma, sigma, sizeof(float)*N*N, hipMemcpyHostToDevice);
    CHECK_ERR(err);

    dim3 BlockSize(1,1);
    dim3 GridSize(N/BlockSize.x, N/BlockSize.y);
    normCalc<<<GridSize,BlockSize>>>(d_A, d_B, d_mu, d_sigma, N);

    err = hipMemcpy(B, (d_B), sizeof(float)*N*N, hipMemcpyDeviceToHost);
    CHECK_ERR(err);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_mu);
    hipFree(d_sigma);

}
